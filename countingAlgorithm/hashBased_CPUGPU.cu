#include "hip/hip_runtime.h"
#include <iostream>
#include <hipcub/hipcub.hpp>
#include <cub/util_type.cuh>
#include<bitset>

__global__ 
void hashBasedButterflyCounting_CPUGPU(int *directNB,long long *par_beginPos, int *edgeList, long long edge_num,long long edge_addr,int uCount, int vCount, unsigned long long* globalCount,  int* hashTable, int startVertex, int endVertex)
{
    __shared__ unsigned long long sharedCount;
    if (threadIdx.x==0) sharedCount=0;
}