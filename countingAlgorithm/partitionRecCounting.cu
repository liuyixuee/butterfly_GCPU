#include "hip/hip_runtime.h"
#include <iostream>
#include <hipcub/hipcub.hpp>
#include <cub/util_type.cuh>
#include<bitset>

using std::bitset;
using namespace std;

__device__ int* binarySearch_2(int* a, int* b, int x)
{
    while (a<b)
    {
        int* mid=a+((b-a)/2);
        if (*mid<=x) a=mid+1; else b=mid;
    }
    return a;
}

__global__
void test_PairCounting(int v_i,int v_j,int *beginPos, int *edgeList,int startId,int edge_n,int vertex_n,int uCount,int vCount,unsigned int *count)
{

    int v_i_end=beginPos[v_i+1]-1;
    int v_j_end;
    if(v_j==vertex_n-1)
    {
        v_j_end=edge_n-1;
    }
    else
    {
        v_j_end=beginPos[v_j+1]-1;
    }

    int p_i=v_i_end,p_j=v_j_end;
    int large_pop=uCount+vCount;
    while(p_i>=0 &&p_j>=0)
    {
        //w>u and w>v
        if(edgeList[p_i]<=v_i+startId)
        {
            p_i=-1;
            break;
        }
        if(edgeList[p_i]>=edgeList[p_j])
        {
            if(edgeList[p_i]==large_pop) *count+=1;
            //if(edgeList[p_i]==large_pop) printf("large_pop is %d, same with edgelist i,there are %d wedges\n",large_pop,*count);
            large_pop=edgeList[p_i];
            p_i--;
        }
        else{
            if(edgeList[p_j]==large_pop) *count+=1;
            //if(edgeList[p_j]==large_pop) printf("large_pop is %d, same with edgelist j\n",large_pop);
            large_pop=edgeList[p_j];
            p_j--;
        }

    }
    if(p_i==-1 &&edgeList[p_j]==large_pop) *count+=1;
    if(p_j==-1 &&edgeList[p_i]==large_pop) *count+=1;


}
__device__
void PairCounting(int v_i,int v_j,int *beginPos, int *edgeList,int startId,int edge_n,int vertex_n,unsigned int *count,int vertexCount)
{

    int v_i_end=beginPos[v_i+1]-1;
    int v_j_end;
    if(v_j==vertex_n-1)
    {
        v_j_end=edge_n-1;
    }
    else
    {
        v_j_end=beginPos[v_j+1]-1;
    }

    int p_i=v_i_end,p_j=v_j_end;
    int large_pop=vertexCount;
    for(;p_i>=beginPos[v_i]&&p_j>=beginPos[v_j];)
    {
        //w>u and w>v
        if(edgeList[p_i]<=v_i+startId)
        {
            p_i=-1;
            break;
        }
        if(edgeList[p_i]>=edgeList[p_j])
        {
            if(edgeList[p_i]==large_pop) *count+=1;
            //if(edgeList[p_i]==large_pop) printf("large_pop is %d, same with edgelist i,there are %d wedges\n",large_pop,*count);
            large_pop=edgeList[p_i];
            //printf("%d==%d ",large_pop,edgeList[p_i]);
            p_i--;
        }
        else{
            if(edgeList[p_j]==large_pop) *count+=1;
            //if(edgeList[p_j]==large_pop) printf("large_pop is %d, same with edgelist j\n",large_pop);
            large_pop=edgeList[p_j];
            //printf("%d==%d ",large_pop,edgeList[p_j]);
            p_j--;
        }

    }
    if(p_i<beginPos[v_i] &&edgeList[p_j]==large_pop) *count+=1;
    if(p_j<beginPos[v_j] &&edgeList[p_i]==large_pop) *count+=1;
    //printf("wedges between %d and %d is: %d\n",v_i,v_j,*count);
    //printf("\n");
}

__global__
void Intra_Partition_Counting(int *beginPos, int *edgeList, int uCount, int vCount,   int* hashTable, int startId,int vertex_n,int edge_n)
{
    unsigned int count=0;
    int vertexCount=uCount+vCount;
    if(blockIdx.x==0 and threadIdx.x==0)
    {
        printf("In this partition vn=%d and en= %d\n",vertex_n,edge_n);
    }
    for(int v_i=0+blockIdx.x;v_i<vertex_n;v_i+=gridDim.x)//a block for a v_i
    {
        for(int v_j=v_i+1+threadIdx.x;v_j<vertex_n;v_j+=blockDim.x)
        {
            PairCounting(v_i,v_j,beginPos,edgeList,startId,edge_n,vertex_n,&count,vertexCount);
            hashTable[v_i*vertex_n+v_j]=count;
            count=0;
        }
        
        
    }
    __syncthreads();
}


__device__
void Inter_PairCounting(int v_i,int v_j,int *beginPos_i,int *beginPos_j,int *edgeList_i,int *edgeList_j,int startId_i,int startId_j,int edge_n_i,int edge_n_j,int vertex_n_i,int vertex_n_j,int uCount,int vCount, unsigned int *count)
{
    
    int v_i_end=v_i==vertex_n_i-1?edge_n_i-1:beginPos_i[v_i+1]-1;//last one differently
    int v_j_end=v_j==vertex_n_j-1?edge_n_j-1:beginPos_j[v_j+1]-1;

    int p_i=v_i_end,p_j=v_j_end;
    int large_pop=uCount+vCount;
    for(;p_i>=beginPos_i[v_i]&&p_j>=beginPos_j[v_j];)
    {
        //w>u and w>v
        if(edgeList_i[p_i]<=v_i+startId_i)
        {
            p_i=-1;
            break;
        }
        if(edgeList_i[p_i]>=edgeList_j[p_j])
        {
            if(edgeList_i[p_i]==large_pop) *count+=1;
            large_pop=edgeList_i[p_i];
            p_i--;
        }
        else{
            if(edgeList_j[p_j]==large_pop) *count+=1;
            large_pop=edgeList_j[p_j];
            p_j--;
        }

    }
    if(p_i<beginPos_i[v_i] &&edgeList_j[p_j]==large_pop) *count+=1;
    if(p_j<beginPos_j[v_j] &&edgeList_i[p_i]==large_pop) *count+=1;
}

__global__ 
void Inter_Partition_Counting(int *beginPos_i,int *beginPos_j, int *edgeList_i, int *edgeList_j, int uCount, int vCount,  int* hashTable, int startId_i,int startId_j,int vertex_n_i,int vertex_n_j,int edge_n_i,int edge_n_j)
{
    unsigned int count=0;
    if(blockIdx.x==0 && threadIdx.x==0)
    {
        printf("In this inter partitions vn_i=%d and en_i= %d   vn_j=%d and en_j= %d\n",vertex_n_i,edge_n_i,vertex_n_j,edge_n_j);
    }
    for(int v_i=0+blockIdx.x;v_i<vertex_n_i;v_i+=gridDim.x)//a block for a v_i
    {
        for(int v_j=0+threadIdx.x;v_j<vertex_n_j;v_j+=blockDim.x)
        {
            
            if(v_j+startId_j>v_i+startId_i) 
            {
                Inter_PairCounting(v_i,v_j,beginPos_i,beginPos_j,edgeList_i,edgeList_j,startId_i,startId_j,edge_n_i,edge_n_j,vertex_n_i,vertex_n_j,uCount,vCount,&count);
                
            }
            else
            {
                Inter_PairCounting(v_j,v_i,beginPos_j,beginPos_i,edgeList_j,edgeList_i,startId_j,startId_i,edge_n_j,edge_n_i,vertex_n_j,vertex_n_i,uCount,vCount,&count);
                
            }
            hashTable[v_i*vertex_n_j+v_j]=count;
            
            count=0;
            
        }
    }
    
    __syncthreads();
}