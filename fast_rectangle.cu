#include <iostream>
#include "graph.h"
#include "wtime.h"
#include "util.h"
#include <hipcub/hipcub.hpp>
#include <fstream>
#include<vector>
#include <cub/util_type.cuh>
#include "countingAlgorithm/partitionRecCounting.cuh"

#define blocknumber 128
#define blocksize 1024
#define hash_blocksize 1024
using namespace std;

int FR(graph* G,int bound )
{
    hipSetDeviceFlags (hipDeviceMapHost);//启用zerocopy
    
    long long vertexCount=G->uCount+G->vCount;
    int *hashTable;
    
    float *time;
    unsigned long long butterfly_num=0;
    vector<int> par_vertex_id;//each partition.i start from par_vertex_id[i]


    //partition
    int par_id=0;
    long long par_sum=0;
    par_vertex_id.push_back(0);
    if(vertexCount>1)
    {
        par_sum=G->beginPos[1]-G->beginPos[0];
    }
    else{
        cout<<"ERROR: less than one vertex"<<endl;
    }
    
    for(int vertex=1;vertex<G->uCount+G->vCount;vertex++)
    {
        long long deg=G->beginPos[vertex+1]-G->beginPos[vertex];
        if(par_sum+deg+1>bound)//是否考虑deg>bound
        {
            par_vertex_id.push_back(vertex);
            cout<<"push back vertex "<<vertex<<endl;
            par_sum=deg+1;
        } 
        else
        {
            par_sum+=(deg+1);
        }
    }
    par_vertex_id.push_back(G->uCount+G->vCount);
    long long largest_par=0;
    for(int i=0;i<par_vertex_id.size()-1;i++)
    {
        if(par_vertex_id[i+1]-par_vertex_id[i]>largest_par) largest_par=par_vertex_id[i+1]-par_vertex_id[i];
    }
    hipHostAlloc((void**) &hashTable, largest_par*largest_par*sizeof(int),
         hipHostMallocWriteCombined | hipHostMallocMapped);
     memset(hashTable, 0, largest_par*largest_par*sizeof(int));
     
     int *D_hashTable;
     hipHostGetDevicePointer(&D_hashTable, hashTable, 0);
    
    //intra_partition
    //Memory Allocating and Data transferring
    

    for(int par_id=0;par_id<par_vertex_id.size()-1;par_id++)
    {
        //definition
        
        int* D_beginPos,*H_beginPos;
        int* D_edgeList;
        int vertex_n=par_vertex_id[par_id+1]-par_vertex_id[par_id];
        int edge_n=G->beginPos[par_vertex_id[par_id+1]]-G->beginPos[par_vertex_id[par_id]];
        //memory allocation
        HRR(hipMalloc(&D_beginPos,sizeof(int)*vertex_n));
        HRR(hipMalloc(&D_edgeList,sizeof(int)*edge_n));
        H_beginPos=new int[vertex_n];
        //initial H_beginPos long long ->int
        for(int v=par_vertex_id[par_id];v<par_vertex_id[par_id+1];v++)
        {
            H_beginPos[v-par_vertex_id[par_id]]=G->beginPos[v]-G->beginPos[par_vertex_id[par_id]];
        }
        
        HRR(hipMemcpy(D_beginPos,H_beginPos,sizeof(int)*vertex_n,hipMemcpyHostToDevice));
        HRR(hipMemcpy(D_edgeList,G->edgeList+G->beginPos[par_vertex_id[par_id]],sizeof(int)*edge_n,hipMemcpyHostToDevice));
        Intra_Partition_Counting<<<blocknumber,hash_blocksize>>>(D_beginPos,D_edgeList,G->uCount,G->vCount,D_hashTable,par_vertex_id[par_id],vertex_n,edge_n);
        HRR(hipDeviceSynchronize());
        HRR(hipFree(D_beginPos));
        HRR(hipFree(D_edgeList));
        for(int i=0;i<vertex_n*vertex_n;i++)
        {
            int ht=hashTable[i];
            butterfly_num+=(ht*(ht-1)/2);
        }
        memset(hashTable, 0, largest_par*largest_par*sizeof(int));
        
    }
    cout<<"intrra butterfly num="<<butterfly_num<<endl;
    //inter_partition
    for(int par_i=0;par_i<par_vertex_id.size()-1;par_i++)
    {
        for(int par_j=par_i+1;par_j<par_vertex_id.size()-1;par_j++)
        {
            printf("iiiii");
            int *beginPos_i,*beginPos_j,*H_beginPos_i,*H_beginPos_j;
            int *edgeList_i,*edgeList_j;
            int vertex_n_i=par_vertex_id[par_i+1]-par_vertex_id[par_i];
            int vertex_n_j=par_vertex_id[par_j+1]-par_vertex_id[par_j];
            int edge_n_i=G->beginPos[par_vertex_id[par_i+1]]-G->beginPos[par_vertex_id[par_i]];
            int edge_n_j=G->beginPos[par_vertex_id[par_j+1]]-G->beginPos[par_vertex_id[par_j]];
            HRR(hipMalloc(&beginPos_i,sizeof(int)*vertex_n_i));
            HRR(hipMalloc(&beginPos_j,sizeof(int)*vertex_n_j));
            HRR(hipMalloc(&edgeList_i,sizeof(int)*edge_n_i));
            HRR(hipMalloc(&edgeList_j,sizeof(int)*edge_n_j));
            H_beginPos_i=new int[vertex_n_i];
            H_beginPos_j=new int[vertex_n_j];
            for(int v=par_vertex_id[par_i];v<par_vertex_id[par_i+1];v++)
            {
                H_beginPos_i[v-par_vertex_id[par_i]]=G->beginPos[v]-G->beginPos[par_vertex_id[par_i]];
            }
            for(int v=par_vertex_id[par_j];v<par_vertex_id[par_j+1];v++)
            {
                H_beginPos_j[v-par_vertex_id[par_i]]=G->beginPos[v]-G->beginPos[par_vertex_id[par_j]];
            }
            HRR(hipMemcpy(beginPos_i,H_beginPos_i,sizeof(int)*vertex_n_i,hipMemcpyHostToDevice));
            HRR(hipMemcpy(edgeList_i,G->edgeList+G->beginPos[par_vertex_id[par_i]],sizeof(int)*edge_n_i,hipMemcpyHostToDevice));
            HRR(hipMemcpy(beginPos_j,H_beginPos_j,sizeof(int)*vertex_n_j,hipMemcpyHostToDevice));
            HRR(hipMemcpy(edgeList_j,G->edgeList+G->beginPos[par_vertex_id[par_j]],sizeof(int)*edge_n_j,hipMemcpyHostToDevice));
            printf("222");
            Inter_Partition_Counting<<<blocknumber,hash_blocksize>>>(beginPos_i,beginPos_j,edgeList_i,edgeList_j,G->uCount,G->vCount,D_hashTable,par_vertex_id[par_i],par_vertex_id[par_j],vertex_n_i,vertex_n_j,edge_n_i,edge_n_j);
            HRR(hipDeviceSynchronize());
            HRR(hipFree(beginPos_i));
            HRR(hipFree(edgeList_i));
            HRR(hipFree(beginPos_j));
            HRR(hipFree(edgeList_j));
            for(int i=0;i<vertex_n_i*vertex_n_j;i++)
            {
                int ht=hashTable[i];
                butterfly_num+=(ht*(ht-1)/2);
            }
            memset(hashTable, 0, largest_par*largest_par*sizeof(int));
        }
    }
    
    // for(long long i=0;i<vertexCount*vertexCount;i++)
    // {
    //     int ht=hashTable[i];
    //     butterfly_num+=(ht*(ht-1)/2);
    // }
    cout<<"total butterfly num="<<butterfly_num<<endl;
    hipHostFree(hashTable);
    return 0;
}